#include "hip/hip_runtime.h"
#include <stdint.h>

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#include <stdio.h>

// kernel
__global__ void Process()
{
    // threadIdx.x
    // threadIdx.y
    // threadIdx.z

    // blockIdx.x
    // blockIdx.y
    // blockIdx.z

    // blockDim.x
    // blockDim.y
    // blockDim.z
    int globalIdx = threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    globalIdx+=blockDim.x*blockDim.y*blockDim.z*(blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x);
}


int main()
{

    int width, height, bpp;

    unsigned char * rgb_image = stbi_load("/content/drive/MyDrive/cats_and_dogs_small/oxana.jpeg", &width, &height, &bpp, 1);
 
    if (rgb_image == nullptr){
        printf("errror\n");
    }
    else{
        printf("width: %d\n", width);
        printf("height: %d\n", height);
        printf("bpp: %d\n", bpp);
        printf("Length of array: %lu\n", (long unsigned int)( sizeof(rgb_image) / sizeof(rgb_image[0]) ));
         
        printf("%d\n", rgb_image[0]);
        printf("size: %lu\n", sizeof(rgb_image));
        printf("last %d\n", rgb_image[1041120]);
        for(int i=0; i<=2000000; i++){
            if (rgb_image[i] == 0){
                printf("last is: %d\n", i);
                break;
            }
            //printf("%d\n", rgb_image[i]);
        }
    }

    stbi_image_free(rgb_image);
    printf("This is done on CPU\n");

    //Process<<<dim3(2,2,2),dim3(2,2,2)>>>();
    //hipDeviceSynchronize();
    return 0;
}
